#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>

#define BLOCK_SIZE 16
#define BASE_TYPE int 
#define rows 16
#define cols 16

__global__ void SumMatrix(const BASE_TYPE* A, const BASE_TYPE* B, BASE_TYPE* C)
{
	int i = cols * (blockDim.y * blockIdx.y + threadIdx.y) + blockDim.x * blockIdx.x + threadIdx.x;
	C[i] = A[i] + B[i];
}
// Функция вычисление числа, которое больше
// числа а и кратное числу b
int findNumber(int a, int b)
{
	int mod = a % b;
	if (mod != 0)
	{
		mod = b - mod;
		return a + mod;
	}
	return a;
}

int main()
{
	size_t size = rows * cols * sizeof(BASE_TYPE);
	hipError_t cudaStatus;

	BASE_TYPE h_A[rows][cols] = { 0 };
	BASE_TYPE h_B[rows][cols] = { 0 };
	BASE_TYPE h_C[rows][cols] = { 0 };

	srand(time(0));
	for (int i = 0; i < rows; i++)
	{
		for (int j = 0; j < cols; j++)
		{
			h_A[i][j] = rand() % 10 + 1;
			h_B[i][j] = rand() % 10 + 1;
		}
	}

	for (int k = 0; k < 2; k++)
	{
		for (int i = 0; i < rows; i++)
		{
			for (int j = 0; j < cols; j++)
				k == 0 ? printf("%d ", h_A[i][j]) : printf("%d ", h_B[i][j]);
			printf("\n");
		}
		printf("\n");
	}

	BASE_TYPE* d_A = NULL;
	hipMalloc((void**)&d_A, size);

	BASE_TYPE* d_B = NULL;
	hipMalloc((void**)&d_B, size);

	BASE_TYPE* d_C = NULL;
	hipMalloc((void**)&d_C, size);

	cudaStatus = hipMemcpy(d_A, &h_A, size, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cudaMempcyA failed!");
		return 1;
	}
	cudaStatus = hipMemcpy(d_B, &h_B, size, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "cudaMempcyB failed!");
		return 2;
	}

	dim3 threadsPerBlock = dim3(BLOCK_SIZE, BLOCK_SIZE);
	dim3 blocksPerGrid = dim3(cols / BLOCK_SIZE, rows / BLOCK_SIZE);

	matrixAdd << <blocksPerGrid, threadsPerBlock >> > (d_A, d_B, d_C);

	cudaStatus = hipMemcpy(&h_C, d_C, size, hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "memcpyC failed!");
		return 4;
	}

	for (int i = 0; i < rows; i++)
	{
		for (int j = 0; j < cols; j++)
			printf("%d ", h_C[i][j]);
		printf("\n");
	}
	printf("\n");

	hipFree(d_A);
	hipFree(d_B);
	hipFree(d_C);
	return 0;
}