#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <cmath>
#include <stdio.h>

#define N 1000 

__global__ void DzetaFunction(float* a, float* b)
{
    int i = threadIdx.x;
    a[i] = 1.f / powf(float(i + 1), *b);
}


int main()
{
    float s = 2;
    float a[N]; 
    float* dev_s = 0;
    float* dev_a = 0;
    float sum = 0; 

    hipMalloc((void**)&dev_a, N * sizeof(float));
    hipMalloc((void**)&dev_s, sizeof(float));
    hipMemcpy(dev_a, a, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dev_s, &s, sizeof(float), hipMemcpyHostToDevice);

    ZFunction << <1, N >> > (dev_a, dev_s);

    hipMemcpy(a, dev_a, N * sizeof(float), hipMemcpyDeviceToHost);

    for (int i = 0; i < N; ++i)
    {
        sum += a[i];
    }
    printf("%f\n", sum);

    hipFree(dev_a);
    hipFree(dev_s);
    return 0;
}