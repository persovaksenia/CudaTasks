#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include "cmath"
#include <stdio.h>

#define N 10 

__global__ void Calculate (float* a, float* b, float* c)
{
	int i = threadIdx.x; 
	if (i > N - 1) return; 
	c[i] = __fmul_rn(a[i], b[i]);
}

int main()
{
	float a[N], b[N], c[N];
	float* dev_a, * dev_b, * dev_c;

	for (int i = 0; i < N; i++)
	{
		a[i] = i + 1;
		b[i] = -2;
	}

	hipMalloc((void**)&dev_a, N * sizeof(float));
	hipMalloc((void**)&dev_b, N * sizeof(float));
	hipMalloc((void**)&dev_c, N * sizeof(float));

	hipMemcpy(dev_a, a, N * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, N * sizeof(float), hipMemcpyHostToDevice);

	Calculate << <1, N >> > (dev_a, dev_b, dev_c);

	hipMemcpy(c, dev_c, N * sizeof(float), hipMemcpyDeviceToHost);

	float result = 0;

	for (int i = 0; i < N; i++)
	{
		result += c[i];
	}

	printf("result = %f\n", result);

	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
	return 0;
}