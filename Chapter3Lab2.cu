#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <cmath>
#include <stdio.h>
#define n 1000 

__global__ void Pi (double* a)
{
    int i = threadIdx.x;
    a[i] = std::sqrtf(1.0 - double(i * i) / double(n * n));
}

int main()
{
    double a[n];
    double* p_a;

    hipMalloc((void**)&p_a, n * sizeof(double));
    Pi << <1, n >> > (d_a);

    hipError_t err = hipGetLastError();

    hipMemcpy(a, p_a, n * sizeof(double), hipMemcpyDeviceToHost);

    if (err != hipSuccess)
        printf("%s ", hipGetErrorString(err)); 
    else
    {
        double q = 0;
        for (int i = 0; i < n; ++i) {
            q += a[i];
        }
        printf("pi = %f\n", q * 4 / n);
    }

    hipFree(p_a);
    return 0;
}