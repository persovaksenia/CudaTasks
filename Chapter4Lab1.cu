#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include "cmath"
#include <stdio.h>

#define N 10 

__global__ void Calculate (float* a, float* b, float* c)
{
	int i = threadIdx.x; //индексирование
	if (i > N - 1) return; 	//проверка на выход за пределы массива
	//поэлементное умножение массивов
	c[i] = __fmul_rn(a[i], b[i]);
}

int main()
{
	// выделение памяти под массивы на CPU
	float a[N], b[N], c[N];
	// выделение памяти под массивы для копирования
	// на GPU
	float* dev_a, * dev_b, * dev_c;

	// заполнение массивов
	for (int i = 0; i < N; i++)
	{
		a[i] = i + 1;
		b[i] = -2;
	}

	// выделение памяти под массивы на GPU
	hipMalloc((void**)&dev_a, N * sizeof(float));
	hipMalloc((void**)&dev_b, N * sizeof(float));
	hipMalloc((void**)&dev_c, N * sizeof(float));

	// копирование данных в память GPU
	hipMemcpy(dev_a, a, N * sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(dev_b, b, N * sizeof(float), hipMemcpyHostToDevice);

	Calculate << <1, N >> > (dev_a, dev_b, dev_c);

	hipMemcpy(c, dev_c, N * sizeof(float), hipMemcpyDeviceToHost);

	float result = 0; //переменная для результата

	//подсчёт скалярного произведения
	for (int i = 0; i < N; i++)
	{
		result += c[i];
	}

	//вывод результата
	printf("result = %f\n", result);

	hipFree(dev_a);
	hipFree(dev_b);
	hipFree(dev_c);
	return 0;
}