#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <time.h>
#include <math.h>
#include <vector>


__global__ void MetodMonteKarlo(int* maxPointsCount, int* pointsInCircleCount, double* coordinateX, double* coordinateY)
{
    for (int i = 0; i < *maxPointsCount; ++i)
    {
        double z = (coordinateX[i] * coordinateX[i]) + (coordinateY[i] * coordinateY[i]);

        if (z <= 1)
        {
            ++* pointsInCircleCount;
        }
    }
}

int main()
{
    //переменные на CPU
    const int maxPointsCount = 15000;
    int pointsInCircleCount = 0;
    double pi;
    //переменные на GPU
    int * dev_maxPointsCount, * dev_pointsInCircleCount;
    int size = sizeof(int); // размерность
    double coordinateX[maxPointsCount], * dev_x;
    double coordinateY[maxPointsCount], * dev_y;
    const size_t x_size = sizeof(double) * size_t(maxPointsCount);

    srand(time(NULL));

    for (int i = 0; i < maxPointsCount; i++)
    {
        coordinateX[i] = (double)rand() / RAND_MAX;
        coordinateY[i] = (double)rand() / RAND_MAX;
    }
    // выделение памяти на GPU
    hipMalloc((void**)&dev_maxPointsCount, size);
    hipMalloc((void**)&dev_pointsInCircleCount, size);

    hipMalloc((void**)&dev_x, x_size);
    hipMalloc((void**)&dev_y, x_size);

    //копирование информации с CPU на GPU
    hipMemcpy(dev_maxPointsCount, &maxPointsCount, size, hipMemcpyHostToDevice);
    hipMemcpy(dev_pointsInCircleCount, &pointsInCircleCount, size, hipMemcpyHostToDevice);

    hipMemcpy(dev_x, coordinateX, x_size, hipMemcpyHostToDevice);
    hipMemcpy(dev_y, coordinateY, x_size, hipMemcpyHostToDevice);

    // вызов ядра
    MetodMonteKarlo << < 1, 1 >> > (dev_maxPointsCount, dev_pointsInCircleCount, dev_x, dev_y);

    // копирование результата работы ядра с GPU на CPU
    hipMemcpy(&pointsInCircleCount, dev_pointsInCircleCount, size, hipMemcpyDeviceToHost);

    pi = ((double)pointsInCircleCount / (double)maxPointsCount) * 4.0;
    printf("Pi: %f\n", pi);

    return 0;
}

